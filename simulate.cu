#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

__global__ void matrixVectorMult(hiprandState* state, float* matrices, float* vectors, float* results, int n) {
    int matrixId = blockIdx.x;  // Each block handles one matrix and its corresponding vectors
    int vectorId = threadIdx.x; // Each thread handles one vector
    hiprandState localState = state[matrixId * 1000 + vectorId]; // Unique state for each vector

    // Pointers to the matrix for this block
    float* matrix = matrices + matrixId * n * 2 * n;
    float* vector = vectors + (matrixId * 1000 + vectorId) * 2 * n;
    float* result = results + (matrixId * 1000 + vectorId) * n;

    // Generate the vector
    for (int i = 0; i < 2 * n; i++) {
        vector[i] = hiprand_uniform(&localState);  // Fill the vector with random numbers
    }

    // Matrix-vector multiplication
    for (int row = 0; row < n; row++) {
        float sum = 0.0;
        for (int col = 0; col < 2 * n; col++) {
            sum += matrix[row * 2 * n + col] * vector[col];
        }
        result[row] = sum;
    }

    // Store the RNG state back
    state[matrixId * 1000 + vectorId] = localState;
}

// Kernel to initialize matrices
__global__ void initializeMatrices(hiprandState* state, float* matrices, int n) {
    int id = blockIdx.x;  // One block per matrix
    hiprandState localState = state[id];  // Unique state per matrix

    float* matrix = matrices + id * n * 2 * n;

    for (int i = 0; i < n * 2 * n; i++) {
        matrix[i] = hiprand_uniform(&localState);  // Fill the matrix with random numbers
    }

    state[id] = localState;
}

int main() {
    int n = 1024;  // dimension
    int num_matrices = 1000;
    int num_vectors_per_matrix = 1000;
    int total_vectors = num_matrices * num_vectors_per_matrix;

    // Allocate memory
    float *d_matrices, *d_vectors, *d_results;
    hipMalloc(&d_matrices, num_matrices * n * 2 * n * sizeof(float)); //n by 2n
    hipMalloc(&d_vectors, total_vectors * 2 * n * sizeof(float)); // 2n
    hipMalloc(&d_results, total_vectors * n * sizeof(float)); // let's be clever and get rid of this
    hiprandState *d_states;
    hipMalloc(&d_states, total_vectors * sizeof(hiprandState)); //vector rng

    // Initialize matrices
    int threadsPerBlock = 1;  // One thread per block for matrix initialization
    initializeMatrices<<<num_matrices, threadsPerBlock>>>(d_states, d_matrices, n);

    setup_states<<<total_vectors, 1>>>(d_states, time(NULL));  // Initialize rng states for each vector

    // Perform matrix-vector multiplications
    threadsPerBlock = 1000;  // One thread per vector, per matrix
    matrixVectorMult<<<num_matrices, threadsPerBlock>>>(d_states, d_matrices, d_vectors, d_results, n);

    // Free memory
    hipFree(d_matrices);
    hipFree(d_vectors);
    hipFree(d_results);
    hipFree(d_states);

    return 0;
}

